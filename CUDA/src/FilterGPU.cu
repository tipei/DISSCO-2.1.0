#include "hip/hip_runtime.h"

#include "../../LASS/src/LASS.h"
#include "FilterGPU.h"

#define CUDA_CHECK(call)                                                     \
do {                                                                         \
    hipError_t err = call;                                                  \
    if (err != hipSuccess) {                                                \
        fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__,     \
                hipGetErrorString(err));                                    \
        exit(EXIT_FAILURE);                                                  \
    }                                                                        \
} while (0)

__global__ void BiQuadFilterGPU(float *inputSample, float* outputSample, float a0, float a1, float a2, float b0, float b1, float b2, long sampleSize){
    float ba0=b0/a0, ba1=b1/a0, ba2=b2/a0, ba3=a1/a0, ba4=a2/a0, bb3, bb4;
    int tx = threadIdx.x, idx;
    __shared__ float Z0[1025], Z1[1025];
    float *Zsrc=Z0, *Zdest=Z1, *Ztemp;
    long ps=(double)(sampleSize+1023)/1024;

    if(threadIdx.x==0){
        Zsrc[threadIdx.x+1]=ba0*inputSample[threadIdx.x];
        Zsrc[0]=0;
        Zdest[0]=0;
    }
    else if(threadIdx.x==1){
        Zsrc[threadIdx.x+1]=ba0*inputSample[threadIdx.x]+ba1*inputSample[threadIdx.x-1];
    }
    else
        Zsrc[threadIdx.x+1]=ba0*inputSample[threadIdx.x]+ba1*inputSample[threadIdx.x-1]+ba2*inputSample[threadIdx.x-2];

    for (int j = 0; j < ps; j++){
        bb3=ba3;
        bb4=ba4;
        for (int off = 1; off < pb*blockDim.x; off *= 2) {
            __syncthreads();
            if (tx >= off) {
                Zdest[tx+1] = Zsrc[tx+1]+bb3*Zsrc[tx+1 - off]+bb4*Zsrc[tx - off];
            }
            else 
                Zdest[tx+1] = Zsrc[tx+1];

            bb3*=bb3;
            bb4*=bb4;
            Ztemp=Zsrc;
            Zsrc=Zdest;
            Zdest=Ztemp;
        }
        outputSample[tx]=Zsrc[tx+1];

        tx+=256;

        if(threadIdx.x==0){
            Zsrc[0]=Zsrc[1023];
            Zdest[0]=Zsrc[1023];
        }
        if(tx<sampleSize)
            Zsrc[threadIdx.x+1]=ba0*inputSample[tx]+ba1*inputSample[tx-1]+ba2*inputSample[tx-2];
        else
            Zsrc[threadIdx.x+1]=0;
        }
}

__global__ void LPCombFilterGPU(float *inputSample, float* outputSample, float inputGain, long inputDelay, float inputLpf_gain, long sampleSize){
    float gain=inputGain, lpf_gain=inputLpf_gain;
    double gaine;
    int tx = threadIdx.x, idx;
    long delay = inputDelay, ps=(double)(sampleSize+delay-1)/delay, pb=(double)(delay+blockDim.x-1)/blockDim.x;
    __shared__ float Z0[4096], Z1[4096];
    float *Zsrc=Z0, *Zdest=Z1, *Ztemp;

   for (int i = 0; i < pb; i++){
        idx = i*blockDim.x + tx;
        if (idx < delay)
            outputSample[idx] = 0;
    }

    for (int i = 0; i < pb; i++){
        idx = i*blockDim.x + tx;
        if (idx < delay){
            Zsrc[idx] = inputSample[idx];
            outputSample[idx+delay] = Zsrc[idx];
        }
    }

    for(int j=2; j<ps; ++j){
        gaine=lpf_gain;
        for (int off = 1; off < pb*blockDim.x; off *= 2) {
            __syncthreads();
            for (int i = 0; i < pb; i++) {
                idx=tx+i*blockDim.x;
                if (idx >= off) {
                    Zdest[idx] = Zsrc[idx]+gaine*Zsrc[idx - off];
                }
                else 
                    Zdest[idx] = Zsrc[idx];
            }
            gaine*=gaine;
            Ztemp=Zsrc;
            Zsrc=Zdest;
            Zdest=Ztemp;
        }

        __syncthreads();
        
        for (int i = 0; i < pb; ++i){
            idx = i*blockDim.x + tx;
            if (idx < delay){
                Zsrc[idx] = gain*Zsrc[idx] + inputSample[(j-1)*delay+idx];
                outputSample[j*delay+idx] = Zsrc[idx];
            }
        }
        if (tx == 0)
            Zsrc[0] +=outputSample[j*delay-1];
    }
}

__global__ void HexAllPassFilterGPU(float *inputSample, float *inputSample0, float *inputSample1, float *inputSample2, float *inputSample3, float *inputSample4, float *inputSample5, float* outputSample, float* envData, float inputGain, long inputDelay, long sampleSize){
    float gain=inputGain, gsqrd=gain*gain, x;
    int tx = blockIdx.x*blockDim.x+threadIdx.x, idx;
    long delay = inputDelay, ps=(double)(sampleSize+delay-1)/delay, pb=(double)(delay+gridDim.x*blockDim.x-1)/(gridDim.x*blockDim.x);
    __shared__ float X[4096], Y[4096];

    for (int i = 0; i < pb; ++i){
        idx = i * gridDim.x * blockDim.x + tx;
        if (idx < delay){
            X[blockDim.x*i+threadIdx.x] = (inputSample0[idx]+inputSample1[idx]+inputSample2[idx]+inputSample3[idx]+inputSample4[idx]+inputSample5[idx])/6;
            Y[blockDim.x*i+threadIdx.x] = -gain*X[blockDim.x*i+threadIdx.x];
            outputSample[idx] = Y[blockDim.x*i+threadIdx.x]*envData[idx] + (1-envData[idx])*inputSample[idx];
        }
    }

    for(int i=1; i<ps; ++i){
        for (int j = 0; j < pb; ++j){
            idx = j* gridDim.x * blockDim.x + tx;
            if (idx < delay&& i*delay+idx < sampleSize){
                x=X[blockDim.x*j+threadIdx.x];
                X[blockDim.x*j+threadIdx.x] = (inputSample0[i*delay+idx]+inputSample1[i*delay+idx]+inputSample2[i*delay+idx]+inputSample3[i*delay+idx]+inputSample4[i*delay+idx]+inputSample5[i*delay+idx])/6;
                Y[blockDim.x*j+threadIdx.x] = -gain*X[blockDim.x*j+threadIdx.x]+(1-gsqrd)*(gain*Y[blockDim.x*j+threadIdx.x]+x);
                outputSample[i*delay+idx] = Y[blockDim.x*j+threadIdx.x]*envData[i*delay+idx]+(1-envData[i*delay+idx])*inputSample[i*delay+idx];
            }
        }
    }
}

__global__ void AllPassFilterGPU(float *inputSample, float* outputSample, float inputGain, long inputDelay, long sampleSize){
    float gain=inputGain, gsqrd=gain*gain, x;
    int tx = threadIdx.x, idx;
    long delay = inputDelay, ps=(double)(sampleSize+delay-1)/delay, pb=(double)(delay+blockDim.x-1)/blockDim.x;
    __shared__ float X[4096], Y[4096];

    for (int i = 0; i < pb; ++i){
        idx = i*blockDim.x + tx;
        if (idx < delay){
            X[blockDim.x*i+threadIdx.x] = inputSample[idx];
            Y[blockDim.x*i+threadIdx.x] = -gain*X[blockDim.x*i+threadIdx.x];
            outputSample[idx] = Y[blockDim.x*i+threadIdx.x] + (1-gain)*X[blockDim.x*i+threadIdx.x];
        }
    }

    for(int i=1; i<ps; ++i){
        for (int j = 0; j < pb; ++j){
            idx = j*blockDim.x + tx;
            if (idx < delay&& i*delay+idx < sampleSize){
                x=X[blockDim.x*j+threadIdx.x];
                X[blockDim.x*j+threadIdx.x] = inputSample[i*delay+idx];
                Y[blockDim.x*j+threadIdx.x] = -gain*X[blockDim.x*j+threadIdx.x]+(1-gsqrd)*(gain*Y[blockDim.x*j+threadIdx.x]+x);
                outputSample[i*delay+idx] = Y[blockDim.x*j+threadIdx.x]+(1-gain)*X[blockDim.x*j+threadIdx.x];
            }
        }
    }
}

__global__ void getEnvData(float *xyPoints, int *segmentTypes, float *envData, int segmentSize, long sampleSize){
    int tx = blockDim.x*blockIdx.x+threadIdx.x, samples, idx, start, i, j;
    float x0, y0, x1, y1, m0=0, m1=0, t, a, b;
    for(i=0; i<segmentSize; ++i){
        x0=xyPoints[2*i];
        y0=xyPoints[2*i+1];
        x1=xyPoints[2*i+2];
        y1=xyPoints[2*i+3];
        samples=(x1-x0)*sampleSize;
        start=sampleSize * x0;
        if(segmentTypes[i]==2){

            m0=(y1-y0)/(x1-x0)/sampleSize;
            for(j=0; j<samples/(blockDim.x*gridDim.x); ++j){
                idx = j * (blockDim.x*gridDim.x)+ tx;
                envData[start + idx] = y0 + idx*m0;
            }

            idx = j * (blockDim.x*gridDim.x) + tx;
            if(start+ idx<sampleSize)
                envData[start + idx] = y0 + idx*m0;

        }
        else if(segmentTypes[i]==1){
            //Cubic Hermite spline
            if(i!=0){
                if(segmentTypes[i-1]==0){
                    a=y0>y1?3:-3;
                    b=pow(2.71828, (double)a);
                    m0=a*(y0-xyPoints[2*i-1])*b/(x0-xyPoints[2*i-2])/(b-1);
                }
                else if(segmentTypes[i-1]==1)
                    m0=(y1-xyPoints[2*i-1])/(x1-xyPoints[2*i-2]);
                else 
                    m0=(y0-xyPoints[2*i-1])/(x0-xyPoints[2*i-2]);
            }
            else
                m0=0;

            if(i+1<segmentSize){
                if(segmentTypes[i+1]==0){
                    a=xyPoints[2*i+5]>y1?3:-3;
                    m1=a*(xyPoints[2*i+5]-y1)/(xyPoints[2*i+4]-x1)/(pow(2.71828, (double)a)-1);
                }
                else if(segmentTypes[i+1]==1)
                    m1=(xyPoints[2*i+5]-y0)/(xyPoints[2*i+4]-x0);
                else 
                    m1=(xyPoints[2*i+5]-y1)/(xyPoints[2*i+4]-x1);
            }
            else
                m1=0;

            a= -3*y0+3*y1-2*m0-m1;
            b= 2*y0-2*y1+m0+m1;

            for(j=0; j<samples/(blockDim.x*gridDim.x); ++j){
                idx = j * (blockDim.x*gridDim.x) + tx;
                t=(float)idx/samples;
                envData[start + idx] = y0 + m0*t + a*t*t + b*t*t*t;
            }
            idx = j * (blockDim.x*gridDim.x) + tx;
            if(start + idx<sampleSize){
                t=(float)idx/samples;
                envData[start + idx] = y0 + m0*t + a*t*t + b*t*t*t;
            }
        }
        else{
            a=y1>y0?3:-3;
            b=pow(2.71828, (double)a);

            for(j=0; j<samples/(blockDim.x*gridDim.x); ++j){
                idx = j * (blockDim.x*gridDim.x) + tx;
                envData[start + idx] = y0+(y1-y0)*(1-pow(2.71828, (double)a*idx/samples))/(1-b);
            }

            idx = j * (blockDim.x*gridDim.x) + tx;
            if(start+ idx<sampleSize)
                envData[start + idx] = y0+(y1-y0)*(1-pow(2.71828, (double)a*idx/samples))/(1-b);
        }
        __syncthreads();
    }
}

void plotWithGnuplot(const std::vector<float>& data) {
    FILE *gnuplotPipe = popen("gnuplot -persistent", "w");
    if (gnuplotPipe) {
        // Set up the plot
        fprintf(gnuplotPipe, "set title 'Plot of Floats from 0 to 2'\n");
        fprintf(gnuplotPipe, "set xlabel 'Index'\n");
        fprintf(gnuplotPipe, "set ylabel 'Value'\n");
        fprintf(gnuplotPipe, "plot '-' with lines\n");
        
        // Send data to GNUplot
        for(size_t i = 0; i < data.size(); ++i){
            fprintf(gnuplotPipe, "%zu %f\n", i, data[i]);
        }
        fprintf(gnuplotPipe, "e\n");
        pclose(gnuplotPipe);
    } else {
        std::cerr << "Could not open pipe to GNUplot.\n";
    }
}

SoundSample* do_reverb_SoundSample_GPU(SoundSample *inWave, Envelope *percentReverbinput, LPCombFilter **lpCombFilter, AllPassFilter *allPassFilter){ 
    SoundSample *outWave=new SoundSample(inWave->getSampleCount(),inWave->getSamplingRate());
    float *inWaveData=inWave->getData(), *outWaveDataD0, *outWaveDataD1, *outWaveDataD2, *outWaveDataD3, *outWaveDataD4, *outWaveDataD5, *outWaveDataD, *inWaveDataD, *outWaveData=new float[inWave->getSampleCount()];
    long sampleSize=inWave->getSampleCount();
    float durationofEnv=percentReverbinput->getDuration();
    float *envData=new float[sampleSize], *envDataD, *envXY, *envXYD;
    int *envSegType, *envSegTypeD, segSize;

    Collection<envelope_segment> *segs=percentReverbinput->getSegments();
    envelope_segment seg;
    segSize=segs->size();

    envXY=new float[segSize*2];
    envSegType=new int[segSize-1];

    for (int i = 1; i < segSize; ++i){
        seg=segs->get(i);
        envXY[i*2]=seg.x;
        envXY[i*2+1]=seg.y;
        envSegType[i-1]=seg.interType;
    }
    
    seg=segs->get(0);
    envXY[0]=seg.x;
    envXY[1]=seg.y;

    hipMalloc(&inWaveDataD, sampleSize*sizeof(float));
    hipMalloc(&outWaveDataD0, sampleSize*sizeof(float));
    hipMalloc(&outWaveDataD1, sampleSize*sizeof(float));
    hipMalloc(&outWaveDataD2, sampleSize*sizeof(float));
    hipMalloc(&outWaveDataD3, sampleSize*sizeof(float));
    hipMalloc(&outWaveDataD4, sampleSize*sizeof(float));
    hipMalloc(&outWaveDataD5, sampleSize*sizeof(float));
    hipMalloc(&outWaveDataD, sampleSize*sizeof(float));
    hipMalloc(&envDataD, sampleSize*sizeof(float));
    hipMemcpy(inWaveDataD, inWaveData, sampleSize*sizeof(float), hipMemcpyHostToDevice);
    LPCombFilterGPU<<<1, 256>>>(inWaveDataD, outWaveDataD0, lpCombFilter[0]->get_g(), lpCombFilter[0]->get_D(), lpCombFilter[0]->get_lpf_g(), sampleSize);
    LPCombFilterGPU<<<1, 256>>>(inWaveDataD, outWaveDataD1, lpCombFilter[1]->get_g(), lpCombFilter[1]->get_D(), lpCombFilter[1]->get_lpf_g(), sampleSize);
    LPCombFilterGPU<<<1, 256>>>(inWaveDataD, outWaveDataD2, lpCombFilter[2]->get_g(), lpCombFilter[2]->get_D(), lpCombFilter[2]->get_lpf_g(), sampleSize);
    LPCombFilterGPU<<<1, 256>>>(inWaveDataD, outWaveDataD3, lpCombFilter[3]->get_g(), lpCombFilter[3]->get_D(), lpCombFilter[3]->get_lpf_g(), sampleSize);
    LPCombFilterGPU<<<1, 256>>>(inWaveDataD, outWaveDataD4, lpCombFilter[4]->get_g(), lpCombFilter[4]->get_D(), lpCombFilter[4]->get_lpf_g(), sampleSize);
    LPCombFilterGPU<<<1, 256>>>(inWaveDataD, outWaveDataD5, lpCombFilter[5]->get_g(), lpCombFilter[5]->get_D(), lpCombFilter[5]->get_lpf_g(), sampleSize);
    hipDeviceSynchronize();

    hipMalloc(&envXYD, segSize*2*sizeof(float));
    hipMalloc(&envSegTypeD, (segSize-1)*sizeof(int));

    hipMemcpy(envXYD, envXY, segSize*2*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(envSegTypeD, envSegType, (segSize-1)*sizeof(int), hipMemcpyHostToDevice);

    getEnvData<<<6, 256>>>(envXYD, envSegTypeD, envDataD, segSize-1, sampleSize);
    hipDeviceSynchronize();

    //cout<<"envXY "<<envXY[0]<<" "<<envXY[1]<<" "<<envXY[2]<<" "<<envXY[3]<<" "<<envXY[4]<<" "<<envXY[5]<<endl;
    //cout<<"envSegType "<<envSegType[0]<<" "<<envSegType[1]<<endl;
    //cout<<"segSize "<<segSize<<endl;
    
    //hipMemcpy(envData, envDataD, sampleSize*sizeof(float), hipMemcpyDeviceToHost);
    //std::vector<float> plot;
    //
    //for (int i = 0; i < sampleSize; i+=1000) {
    //    plot.push_back(envData[i]);
    //}
    //plotWithGnuplot(plot);

    HexAllPassFilterGPU<<<6, 256>>>(inWaveDataD, outWaveDataD0, outWaveDataD1, outWaveDataD2, outWaveDataD3, outWaveDataD4, outWaveDataD5, outWaveDataD, envDataD, allPassFilter->get_g(), allPassFilter->get_D(), sampleSize);

    hipDeviceSynchronize();


    hipMemcpy(outWave->getData(), outWaveDataD, sampleSize*sizeof(float), hipMemcpyDeviceToHost);
    cout<<"outwave 0 "<<(*outWave)[0]<<endl;
    cout<<"outwave 1000 "<<(*outWave)[1000]<<endl;
    cout<<"outwave 10000 "<<(*outWave)[10000]<<endl;
    cout<<"outwave 100000 "<<(*outWave)[100000]<<endl;

    hipFree(inWaveDataD);
    hipFree(outWaveDataD0);
    hipFree(outWaveDataD1);
    hipFree(outWaveDataD2);
    hipFree(outWaveDataD3);
    hipFree(outWaveDataD4);
    hipFree(outWaveDataD5);
    hipFree(outWaveDataD);
    hipFree(envDataD);
    hipFree(envXYD);
    hipFree(envSegTypeD);
    delete[] envData;
    delete[] outWaveData;
    delete[] envXY;
    delete[] envSegType;

    return outWave;
}

SoundSample* do_biquad_filter_GPU(SoundSample *inWave, BiQuadFilter *biQuadFilter){
    SoundSample *outWave=new SoundSample(inWave->getSampleCount(),inWave->getSamplingRate());
    float *inWaveData=inWave->getData(), *outWaveDataD, *inWaveDataD, *outWaveData=new float[inWave->getSampleCount()];
    long sampleSize=inWave->getSampleCount();

    hipMalloc(&inWaveDataD, sampleSize*sizeof(float));
    hipMalloc(&outWaveDataD, sampleSize*sizeof(float));
    hipMemcpy(inWaveDataD, inWaveData, sampleSize*sizeof(float), hipMemcpyHostToDevice);
    BiQuadFilterGPU<<<1, 1024>>>(inWaveDataD, outWaveDataD, biQuadFilter->get_a0(), biQuadFilter->get_a1(), biQuadFilter->get_a2(), biQuadFilter->get_b0(), biQuadFilter->get_b1(), biQuadFilter->get_b2(), sampleSize);

    hipDeviceSynchronize();

    hipMemcpy(outWave->getData(), outWaveDataD, sampleSize*sizeof(float), hipMemcpyDeviceToHost);
    cout<<"outwave 0 "<<(*outWave)[0]<<endl;
    cout<<"outwave 1000 "<<(*outWave)[1000]<<endl;
    cout<<"outwave 10000 "<<(*outWave)[10000]<<endl;
    cout<<"outwave 100000 "<<(*outWave)[100000]<<endl;

    hipFree(inWaveDataD);
    hipFree(outWaveDataD);
    delete[] outWaveData;
    
    return outWave;
}

SoundSample* do_all_pass_filter_GPU(SoundSample *inWave, AllPassFilter *allPassFilter){
    SoundSample *outWave=new SoundSample(inWave->getSampleCount(),inWave->getSamplingRate());
    float *inWaveData=inWave->getData(), *outWaveDataD, *inWaveDataD, *outWaveData=new float[inWave->getSampleCount()];
    long sampleSize=inWave->getSampleCount();

    hipMalloc(&inWaveDataD, sampleSize*sizeof(float));
    hipMalloc(&outWaveDataD, sampleSize*sizeof(float));
    hipMemcpy(inWaveDataD, inWaveData, sampleSize*sizeof(float), hipMemcpyHostToDevice);
    AllPassFilterGPU<<<6, 256>>>(inWaveDataD, outWaveDataD, allPassFilter->get_g(), allPassFilter->get_D(), sampleSize);

    hipDeviceSynchronize();

    hipMemcpy(outWave->getData(), outWaveDataD, sampleSize*sizeof(float), hipMemcpyDeviceToHost);
    cout<<"outwave 0 "<<(*outWave)[0]<<endl;
    cout<<"outwave 1000 "<<(*outWave)[1000]<<endl;
    cout<<"outwave 10000 "<<(*outWave)[10000]<<endl;
    cout<<"outwave 100000 "<<(*outWave)[100000]<<endl;

    hipFree(inWaveDataD);
    hipFree(outWaveDataD);
    delete[] outWaveData;
    
    return outWave;
}